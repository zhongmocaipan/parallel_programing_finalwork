#include "hip/hip_runtime.h"
#include<iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

class Image {
public:
    int width, height;
    std::vector<float> data;

    Image(int w, int h) : width(w), height(h), data(w * h) {}

    float& at(int x, int y) {
        return data[y * width + x];
    }

    const float& at(int x, int y) const {
        return data[y * width + x];
    }
};

// CUDA 内核：水平高斯滤波
__global__ void horizontalBlurKernel(float* input, float* output, int width, int height, float* kernel, int radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        float value = 0.0f;
        for (int k = -radius; k <= radius; ++k) {
            int ix = min(max(x + k, 0), width - 1);
            value += input[y * width + ix] * kernel[k + radius];
        }
        output[y * width + x] = value;
    }
}

// CUDA 内核：垂直高斯滤波
__global__ void verticalBlurKernel(float* input, float* output, int width, int height, float* kernel, int radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        float value = 0.0f;
        for (int k = -radius; k <= radius; ++k) {
            int iy = min(max(y + k, 0), height - 1);
            value += input[iy * width + x] * kernel[k + radius];
        }
        output[y * width + x] = value;
    }
}

// 高斯滤波器 - CUDA 实现
void gaussianBlurCUDA(Image& img, float sigma) {
    int radius = static_cast<int>(std::ceil(3 * sigma));
    std::vector<float> kernel(2 * radius + 1);
    float sum = 0.0f;

    for (int i = -radius; i <= radius; ++i) {
        kernel[i + radius] = std::exp(-(i * i) / (2 * sigma * sigma));
        sum += kernel[i + radius];
    }

    for (auto& k : kernel) {
        k /= sum;
    }

    float *d_input, *d_temp, *d_output, *d_kernel;
    size_t imgSize = img.width * img.height * sizeof(float);
    size_t kernelSize = kernel.size() * sizeof(float);

    // 分配 GPU 内存
    hipMalloc((void)&d_input, imgSize);
    hipMalloc((void)&d_temp, imgSize);
    hipMalloc((void)&d_output, imgSize);
    hipMalloc((void)&d_kernel, kernelSize);

    // 复制数据到 GPU
    hipMemcpy(d_input, img.data.data(), imgSize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.data(), kernelSize, hipMemcpyHostToDevice);

    // 定义 CUDA 网格和块
    dim3 blockSize(16, 16);
    dim3 gridSize((img.width + blockSize.x - 1) / blockSize.x, (img.height + blockSize.y - 1) / blockSize.y);

    // 执行水平高斯滤波
    horizontalBlurKernel<<<gridSize, blockSize>>>(d_input, d_temp, img.width, img.height, d_kernel, radius);
    hipDeviceSynchronize();

    // 执行垂直高斯滤波
    verticalBlurKernel<<<gridSize, blockSize>>>(d_temp, d_output, img.width, img.height, d_kernel, radius);
    hipDeviceSynchronize();

    // 复制结果回主机
    hipMemcpy(img.data.data(), d_output, imgSize, hipMemcpyDeviceToHost);

    // 释放 GPU 内存
    hipFree(d_input);
    hipFree(d_temp);
    hipFree(d_output);
    hipFree(d_kernel);
}

// 高斯差分
Image differenceOfGaussiansCUDA(const Image& img, float sigma1, float sigma2) {
    Image blurred1 = img;
    Image blurred2 = img;
    gaussianBlurCUDA(blurred1, sigma1);
    gaussianBlurCUDA(blurred2, sigma2);

    Image dog(img.width, img.height);
    for (int y = 0; y < img.height; ++y) {
        for (int x = 0; x < img.width; ++x) {
            dog.at(x, y) = blurred1.at(x, y) - blurred2.at(x, y);
        }
    }
    return dog;
}

int main() {
    // 示例图像
    int width = 512;
    int height = 512;
    Image img(width, height);

    // 生成随机图像数据
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            img.at(x, y) = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    // 构建DOG尺度空间
    float sigma1 = 1.0f;
    float sigma2 = 2.0f;
    Image dog = differenceOfGaussiansCUDA(img, sigma1, sigma2);

    // 检测特征点
    std::vector<std::pair<int, int>> keypoints = detectKeypoints(dog);

    // 输出特征点数量
    std::cout << "Detected " << keypoints.size() << " keypoints.\n";

    return 0;
}